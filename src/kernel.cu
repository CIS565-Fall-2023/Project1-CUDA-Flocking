#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"
#include <thrust/device_vector.h>
#include <thrust/scan.h>

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 256

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.05f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f
/*! Size of the grid equals to max rule distance. */
#define SINGLE_MAX_DISTANCE_GRID 0


/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.  

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?
int* dev_gridCellPartitions;
int* dev_gridCellPartitionsPrefixSum;
int* dev_B0start;
int* dev_B0offset;
int B0_size=0;

__device__ unsigned int maxNumParticlesInGrid = 0;

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
glm::vec3* dev_pos_reordered;
glm::vec3* dev_vel1_reordered;
glm::vec3* dev_vel2_reordered;
// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  float maxDistance = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
  // LOOK-2.1 computing grid params
#if SINGLE_MAX_DISTANCE_GRID
  gridCellWidth = 1.0f * maxDistance;
#else
  gridCellWidth = 2.0f * maxDistance;
#endif
  int halfSideCount = (scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");
  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");
  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");
  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");
  dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
  dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
  hipMalloc((void**)&dev_pos_reordered, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos_reordered failed!");
  hipMalloc((void**)&dev_vel1_reordered, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1_reordered failed!");
  hipMalloc((void**)&dev_vel2_reordered, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2_reordered failed!");

  hipMalloc((void**)&dev_gridCellPartitions, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellPartitions failed!");
  hipMalloc((void**)&dev_gridCellPartitionsPrefixSum, gridCellCount * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellPartitionsPrefixSum failed!");

  B0_size = gridCellCount;

  hipMalloc((void**)&dev_B0start, B0_size * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_B0start failed!");
  hipMalloc((void**)&dev_B0offset, B0_size * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_B0offset failed!");

  int nil = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(maxNumParticlesInGrid), &nil, sizeof(int));
  checkCUDAErrorWithLine("hipMemcpyToSymbol maxNumParticlesInGrid failed!");

  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  std::cout << "CUDA device max shared memory per block:" << devProp.sharedMemPerBlock<<std::endl;

  hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
  checkCUDAErrorWithLine("hipDeviceSetSharedMemConfig hipSharedMemBankSizeFourByte failed!");
  hipDeviceSynchronize();

}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
  // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
  // Rule 2: boids try to stay a distance d away from each other
  // Rule 3: boids try to match the speed of surrounding boids
    glm::vec3 percived_center = glm::vec3(0);
    glm::vec3 v = vel[iSelf];
    int num_neighbours = 0;
    glm::vec3 percived_velocity = glm::vec3(0);
    glm::vec3 c = glm::vec3(0);
    for (int i = 0; i < N; i++)
    {
        float dist = glm::distance(pos[i], pos[iSelf]);
        if (i != iSelf && dist < rule1Distance)//rule1Distance==rule3Distance
        {
            num_neighbours++;
            percived_velocity += vel[i];
            percived_center += pos[i];
            if (dist < rule2Distance)
            {
                c -= (pos[i] - pos[iSelf]);
            }
        }
    }
    if (num_neighbours)
    {
        percived_center /= num_neighbours;
        v += (percived_center - pos[iSelf]) * rule1Scale;
        v += percived_velocity * rule3Scale / (float)num_neighbours;
        v += c * rule2Scale;
    }    
  return v;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if (i >= N) {
        return;
    }
    vel2[i] = glm::clamp(computeVelocityChange(N, i, pos, vel1), -maxSpeed, maxSpeed);
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int* indices, int* gridIndices) {
    // TODO-2.1
    // - Label each boid with the index of its grid cell.
    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;
    indices[index] = index;
    glm::vec3 fidx = (pos[index] - gridMin) * inverseCellWidth;
    gridIndices[index] = gridIndex3Dto1D((int)fidx.x, (int)fidx.y, (int)fidx.z, gridResolution);
}

__global__ void kernComputeMortonCodeNaive(int N, int gridResolution,
    glm::vec3 gridMin, float inverseCellWidth,
    glm::vec3* pos, int* indices, uint64_t* zIndices) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;
    indices[index] = index;
    glm::vec3 fidx = (pos[index] - gridMin) * inverseCellWidth;
    uint64_t x = fidx.x, y = fidx.y, z = fidx.z;
    uint64_t zidx = 0;
    for (int i = 0; i < 21; i++)
    {
        zidx |= (x & 1) << (i * 3);
        zidx |= (y & 1) << (i * 3 + 1);
        zidx |= (z & 1) << (i * 3 + 2);
        x >>= 1; y >>= 1; z >>= 1;
    }
    zIndices[index] = zidx;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int* particleGridIndices,
    int* gridCellStartIndices, int* gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;
    if (index == 0)
    {
        gridCellStartIndices[particleGridIndices[index]] = index;
    }
    else
    {
        if (particleGridIndices[index - 1] != particleGridIndices[index])
        {
            gridCellEndIndices[particleGridIndices[index - 1]] = index;
            gridCellStartIndices[particleGridIndices[index]] = index;
        }
        if (index == N - 1)
        {
            gridCellEndIndices[particleGridIndices[index]] = index + 1;
        }
    }
}

__global__ void kernIdentifyMaxNumParticlesAndPartitionsInGrid(int N, int* gridCellStartIndices, int* gridCellEndIndices, int* partitionsForGrid)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    __shared__ int blk_sz;
    if (threadIdx.x == 0) blk_sz = 0;
    __syncthreads();
    if (index < N)
    {
        int localsz = gridCellEndIndices[index] - gridCellStartIndices[index];
        atomicMax(&blk_sz, localsz);
        partitionsForGrid[index] = (localsz + blockSize - 1) / blockSize;
    }
    __syncthreads();
    if(threadIdx.x == 0)
        atomicMax(&maxNumParticlesInGrid, blk_sz);
}

__global__ void kernCompactArray(int N,int* gridCellPartitions, int* gridCellPartitionsPrefixSum,int* B0start,int* B0offset)
{
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) return;
    int partitionSize = gridCellPartitions[index];
    if (partitionSize)
    {
        int b0idx = gridCellPartitionsPrefixSum[index];
        for (int i = 0; i < partitionSize; i++)
        {
            B0start[b0idx + i] = index;
            B0offset[b0idx + i] = i;
        }
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
    int* gridCellStartIndices, int*gridCellEndIndices,
    int* particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
    int selfIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfIndex >= N) return;
  // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
  // the number of boids that need to be checked.
  // - Identify the grid cell that this particle is in
    glm::vec3 fidx = (pos[selfIndex] - gridMin) * inverseCellWidth;
    glm::ivec3 idx = glm::floor(fidx);
    glm::vec3 v = vel1[selfIndex];
    int num_neighbours = 0;
    glm::vec3 percived_velocity = glm::vec3(0);
    glm::vec3 percived_center = glm::vec3(0);
    glm::vec3 c = glm::vec3(0);
    // - Identify which cells may contain neighbors. This isn't always 8.
    // - For each cell, read the start/end indices in the boid pointer array.
    // - Access each boid in the cell and compute velocity change from
    //   the boids rules, if this boid is within the neighborhood distance.
#if SINGLE_MAX_DISTANCE_GRID
    for(int z=-1;z<=1;z++)
        for (int y = -1; y <= 1; y++)
            for (int x = -1; x <= 1; x++)
            {
                int nx = idx.x + x, ny = idx.y + y, nz = idx.z + z;
                if (nx < 0 || nx >= gridResolution || ny < 0 || ny >= gridResolution || nz < 0 || nz >= gridResolution)
                {
                    continue;
                }
                int flattenedCellIdx = gridIndex3Dto1D(nx, ny, nz, gridResolution);
                if (gridCellStartIndices[flattenedCellIdx]>=0)
                {
                    for (int arrayIdx = gridCellStartIndices[flattenedCellIdx]; arrayIdx != gridCellEndIndices[flattenedCellIdx]; arrayIdx++)
                    {
                        int other = particleArrayIndices[arrayIdx];
                        float dist = glm::distance(pos[selfIndex], pos[other]);
                        if (other != selfIndex && dist < rule1Distance)//assume rule1Distance==rule3Distance
                        {
                            num_neighbours++;
                            percived_velocity += vel1[other];
                            percived_center += pos[other];
                            if (dist < rule2Distance)
                            {
                                c -= (pos[other] - pos[selfIndex]);
                            }
                        }
                    }
                }
            }
#else
    glm::vec3 tmp = fidx - glm::floor(fidx);
    int dx = tmp.x > 0.5 ? 1 : -1;
    int dy = tmp.y > 0.5 ? 1 : -1;
    int dz = tmp.z > 0.5 ? 1 : -1;
    for(int z = 0;z < 2; z++)
        for(int y = 0;y < 2; y++)
            for (int x = 0; x < 2; x++)
            {
                int nx = idx.x + x * dx;
                int ny = idx.y + y * dy;
                int nz = idx.z + z * dz;
                if (nx < 0 || nx >= gridResolution || ny < 0 || ny >= gridResolution || nz < 0 || nz >= gridResolution)
                {
                    continue;
                }
                int flattenedCellIdx = gridIndex3Dto1D(nx, ny, nz, gridResolution);
                if (gridCellStartIndices[flattenedCellIdx]>=0)
                {
                    for (int arrayIdx = gridCellStartIndices[flattenedCellIdx]; arrayIdx != gridCellEndIndices[flattenedCellIdx]; arrayIdx++)
                    {
                        int other = particleArrayIndices[arrayIdx];
                        float dist = glm::distance(pos[selfIndex], pos[other]);
                        if (other != selfIndex && dist <= rule1Distance)//assume rule1Distance==rule3Distance
                        {
                            num_neighbours++;
                            percived_velocity += vel1[other];
                            percived_center += pos[other];
                            if (dist <= rule2Distance)
                            {
                                c -= (pos[other] - pos[selfIndex]);
                            }
                        }
                    }
                }
            }
#endif
    if (num_neighbours)
    {
        percived_center /= (num_neighbours);
        v += (percived_center - pos[selfIndex]) * rule1Scale;
        v += percived_velocity * rule3Scale / ((float)num_neighbours);
        v += c * rule2Scale;
    }
  // - Clamp the speed change before putting the new speed in vel2
    vel2[selfIndex] = glm::clamp(v, -maxSpeed, maxSpeed);
}

__global__ void kernShufflePosAndVel1(int N, int* particleArrayIndices, glm::vec3* pos, glm::vec3* vel, glm::vec3* pos_s, glm::vec3* vel_s)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= N) return;
    pos_s[i] = pos[particleArrayIndices[i]];
    vel_s[i] = vel[particleArrayIndices[i]];
}

__global__ void kernUnshuffleVel2(int N, int* particleArrayIndices, glm::vec3* vel2_s, glm::vec3* vel2)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= N) return;
    vel2[particleArrayIndices[i]] = vel2_s[i];
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
    int*gridCellStartIndices, int*gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
    int selfIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfIndex >= N) return;
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
    glm::vec3 fidx = (pos[selfIndex] - gridMin) * inverseCellWidth;
    glm::ivec3 idx = fidx;
    glm::vec3 tmp = fidx - glm::floor(fidx);
    int dx = tmp.x > 0.5 ? 1 : -1, dy = tmp.y > 0.5 ? 1 : -1, dz = tmp.z > 0.5 ? 1 : -1;

    glm::vec3 v = vel1[selfIndex];
    int num_neighbours = 0;
    glm::vec3 percived_velocity = glm::vec3(0);
    glm::vec3 percived_center = glm::vec3(0);
    glm::vec3 c = glm::vec3(0);
  // - Identify which cells may contain neighbors. This isn't always 8.
  // - For each cell, read the start/end indices in the boid pointer array.
  //   DIFFERENCE: For best results, consider what order the cells should be
  //   checked in to maximize the memory benefits of reordering the boids data.
  // - Access each boid in the cell and compute velocity change from
  //   the boids rules, if this boid is within the neighborhood distance.
#if SINGLE_MAX_DISTANCE_GRID
    for (int z = -1; z <= 1; z++)
        for (int y = -1; y <= 1; y++)
            for (int x = -1; x <= 1; x++)
            {
                int nx = idx.x + x, ny = idx.y + y, nz = idx.z + z;
                if (nx < 0 || nx >= gridResolution || ny < 0 || ny >= gridResolution || nz < 0 || nz >= gridResolution)
                {
                    continue;
                }
                int flattenedCellIdx = gridIndex3Dto1D(nx, ny, nz, gridResolution);
                if (gridCellStartIndices[flattenedCellIdx]>=0)
                {
                    for (int other = gridCellStartIndices[flattenedCellIdx]; other != gridCellEndIndices[flattenedCellIdx]; other++)
                    {
                        float dist = glm::distance(pos[selfIndex], pos[other]);
                        if (other != selfIndex && dist < rule1Distance)//assume rule1Distance==rule3Distance
                        {
                            num_neighbours++;
                            percived_velocity += vel1[other];
                            percived_center += pos[other];
                            if (dist < rule2Distance)
                            {
                                c -= (pos[other] - pos[selfIndex]);
                            }
                        }
                    }
                }
            }
#else
    for (int z = 0; z < 2; z++)
        for (int y = 0; y < 2; y++)
            for (int x = 0; x < 2; x++)
            {
                int nx = idx.x + x * dx, ny = idx.y + y * dy, nz = idx.z + z * dz;
                if (nx < 0 || nx >= gridResolution || ny < 0 || ny >= gridResolution || nz < 0 || nz >= gridResolution)
                {
                    continue;
                }
                int flattenedCellIdx = gridIndex3Dto1D(nx, ny, nz, gridResolution);
                if (gridCellStartIndices[flattenedCellIdx] >= 0)
                {
                    for (int other = gridCellStartIndices[flattenedCellIdx]; other != gridCellEndIndices[flattenedCellIdx]; other++)
                    {
                        float dist = glm::distance(pos[selfIndex], pos[other]);
                        if (other != selfIndex && dist < rule1Distance)//assume rule1Distance==rule3Distance
                        {
                            num_neighbours++;
                            percived_velocity += vel1[other];
                            percived_center += pos[other];
                            if (dist < rule2Distance)
                            {
                                c -= (pos[other] - pos[selfIndex]);
                            }
                        }
                    }
                }
            }
#endif
  
    if (num_neighbours)
    {
        percived_center /= num_neighbours;
        v += (percived_center - pos[selfIndex]) * rule1Scale;
        v += percived_velocity * rule3Scale / (float)num_neighbours;
        v += c * rule2Scale;
    }
  // - Clamp the speed change before putting the new speed in vel2
    vel2[selfIndex] = glm::clamp(v, -maxSpeed, maxSpeed);
}

__global__ void kernUpdateVelNeighborSearchCoherentGridLoopingOptimization(
    int N, int gridResolution, glm::vec3 gridMin,
    float inverseCellWidth, float cellWidth,
    int* gridCellStartIndices, int* gridCellEndIndices,
    glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
    int selfIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (selfIndex >= N) return;

    glm::vec3 fidxmi = glm::clamp((pos[selfIndex] - glm::vec3(rule1Distance) - gridMin) * inverseCellWidth, (float)0, (float)gridResolution - 1);
    glm::vec3 fidxmx = glm::clamp((pos[selfIndex] + glm::vec3(rule1Distance) - gridMin) * inverseCellWidth, (float)0, (float)gridResolution - 1);
    glm::ivec3 mi = fidxmi, mx = fidxmx;

    glm::vec3 v = vel1[selfIndex];
    int num_neighbours = 0;
    glm::vec3 percived_velocity = glm::vec3(0);
    glm::vec3 percived_center = glm::vec3(0);
    glm::vec3 c = glm::vec3(0);

    for(int z=mi.z;z<=mx.z;z++)
        for(int y=mi.y;y<=mx.y;y++)
            for (int x = mi.x; x <= mx.x; x++)
            {
                int flattenedCellIdx = gridIndex3Dto1D(x, y, z, gridResolution);
                if (gridCellStartIndices[flattenedCellIdx] == -1) continue;
                for (int other = gridCellStartIndices[flattenedCellIdx]; other != gridCellEndIndices[flattenedCellIdx]; other++)
                {
                    float dist = glm::distance(pos[selfIndex], pos[other]);
                    if (other != selfIndex && dist < rule1Distance)//assume rule1Distance==rule3Distance
                    {
                        num_neighbours++;
                        percived_velocity += vel1[other];
                        percived_center += pos[other];
                        if (dist < rule2Distance)
                        {
                            c -= (pos[other] - pos[selfIndex]);
                        }
                    }
                }
            }

    if (num_neighbours)
    {
        percived_center /= num_neighbours;
        v += (percived_center - pos[selfIndex]) * rule1Scale;
        v += percived_velocity * rule3Scale / (float)num_neighbours;
        v += c * rule2Scale;
    }
    // - Clamp the speed change before putting the new speed in vel2
    vel2[selfIndex] = glm::clamp(v, -maxSpeed, maxSpeed);
}

#define NUM_COPY_THREADS 108

__global__ void kernUpdateVelNeighborSearchCoherentSharedMemoryOptimization(
    int N, int gridResolution,int gridMaxNumParticles, glm::vec3 gridMin,
    float inverseCellWidth, float cellWidth,
    const int* gridCellStartIndices, const int* gridCellEndIndices,const int* b0start, const int* b0offset,
    const glm::vec3* pos, const glm::vec3* vel1, glm::vec3* vel2) {
    int indexInGrid = b0offset[blockIdx.x] * blockSize + threadIdx.x;
    int selfFlattenedGridIdx = b0start[blockIdx.x];
    int particleIdxEnd = gridCellEndIndices[selfFlattenedGridIdx];
    int particleIdxStart = gridCellStartIndices[selfFlattenedGridIdx];
    int gridNumParticles = particleIdxEnd - particleIdxStart;
    
    
    glm::ivec3 gridIdx = glm::ivec3((selfFlattenedGridIdx % (gridResolution * gridResolution)) % gridResolution, (selfFlattenedGridIdx % (gridResolution * gridResolution)) / gridResolution, selfFlattenedGridIdx / (gridResolution * gridResolution));
    int localIndex = threadIdx.x;
    extern __shared__ glm::vec3 s[];
    
    if (localIndex < NUM_COPY_THREADS)
    {
        int w = localIndex / 27;
        int li = localIndex % 27;
        int x = (li % 9) % 3 - 1, y = (li % 9) / 3 - 1, z = li / 9 - 1;
        int nx = gridIdx.x + x, ny = gridIdx.y + y, nz = gridIdx.z + z;
        if (nx >= 0 || nx < gridResolution || ny >= 0 || ny < gridResolution || nz >= 0 || nz < gridResolution)
        {
            int flattenedCellIdx = gridIndex3Dto1D(nx, ny, nz, gridResolution);
            if (gridCellStartIndices[flattenedCellIdx] >= 0)
            {
                for (int other = gridCellStartIndices[flattenedCellIdx] + w, i = w; other < gridCellEndIndices[flattenedCellIdx]; other+= NUM_COPY_THREADS/27,i+= NUM_COPY_THREADS / 27)
                {
                    s[gridMaxNumParticles * li * 2 + i * 2] = pos[other];
                    s[gridMaxNumParticles * li * 2 + i * 2 + 1] = vel1[other];
                }
            }
        }
    }
    __syncthreads();
    if (indexInGrid < gridNumParticles)
    {
        // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
        // except with one less level of indirection.
        // This should expect gridCellStartIndices and gridCellEndIndices to refer
        // directly to pos and vel1.
        // - Identify the grid cell that this particle is in
        int selfIndex = particleIdxStart + b0offset[blockIdx.x] * blockSize + threadIdx.x;
        int blockOffset = indexInGrid * 2;
        int centerGridOffset = gridMaxNumParticles * (27/2) * 2;
        glm::vec3 currPos = s[centerGridOffset + blockOffset];
        glm::vec3 v = s[centerGridOffset + blockOffset + 1];
        int num_neighbours = 0;
        glm::vec3 percived_velocity = glm::vec3(0);
        glm::vec3 percived_center = glm::vec3(0);
        glm::vec3 c = glm::vec3(0);
        // - Identify which cells may contain neighbors. This isn't always 8.
        // - For each cell, read the start/end indices in the boid pointer array.
        //   DIFFERENCE: For best results, consider what order the cells should be
        //   checked in to maximize the memory benefits of reordering the boids data.
        // - Access each boid in the cell and compute velocity change from
        //   the boids rules, if this boid is within the neighborhood distance.
        for (int z = -1; z <= 1; z++)
            for (int y = -1; y <= 1; y++)
                for (int x = -1; x <= 1; x++)
                {
                    int nx = gridIdx.x + x, ny = gridIdx.y + y, nz = gridIdx.z + z;
                    if (nx < 0 || nx >= gridResolution || ny < 0 || ny >= gridResolution || nz < 0 || nz >= gridResolution)
                    {
                        continue;
                    }
                    int flattenedCellIdx = gridIndex3Dto1D(nx, ny, nz, gridResolution);
                    int smCellIdx = gridIndex3Dto1D(x + 1, y + 1, z + 1, 3);
                    if (gridCellStartIndices[flattenedCellIdx] >= 0)
                    {
                        for (int other = gridCellStartIndices[flattenedCellIdx],i=0; other != gridCellEndIndices[flattenedCellIdx]; other++,i++)
                        {
                            float dist = glm::distance(currPos, s[gridMaxNumParticles * smCellIdx * 2 + i * 2]);
                            if (other != selfIndex && dist < rule1Distance)//assume rule1Distance==rule3Distance
                            {
                                num_neighbours++;
                                percived_velocity += s[gridMaxNumParticles * smCellIdx * 2 + i * 2 + 1];
                                percived_center += s[gridMaxNumParticles * smCellIdx * 2 + i * 2];
                                if (dist < rule2Distance)
                                {
                                    c -= (s[gridMaxNumParticles * smCellIdx * 2 + i * 2] - currPos);
                                }
                            }
                        }
                    }
                }

        if (num_neighbours)
        {
            percived_center /= num_neighbours;
            v += (percived_center - currPos) * rule1Scale;
            v += percived_velocity * rule3Scale / (float)num_neighbours;
            v += c * rule2Scale;
        }
        // - Clamp the speed change before putting the new speed in vel2
        vel2[selfIndex] = glm::clamp(v, -maxSpeed, maxSpeed);
    }
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
    int n = (numObjects + blockSize - 1) / blockSize;
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
    kernUpdateVelocityBruteForce << <n, blockSize >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
    kernUpdatePos << <n, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
  // TODO-1.2 ping-pong the velocity buffers
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
    int n1 = (numObjects + blockSize - 1) / blockSize;
    int n2 = (gridCellCount + blockSize - 1) / blockSize;
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
    kernComputeIndices << < n1, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
    kernIdentifyCellStartEnd << < n1, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered << < n1, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
  // - Update positions
    kernUpdatePos << <n1, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
  // - Ping-pong buffers as needed
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
    int n1 = (numObjects + blockSize - 1) / blockSize;
    int n2 = (gridCellCount + blockSize - 1) / blockSize;
    int N = numObjects;
  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
    kernComputeIndices << < n1, blockSize >> > (N, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
    kernIdentifyCellStartEnd << < n1, blockSize >> > (N, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    kernShufflePosAndVel1 << < n1, blockSize >> > (N, dev_particleArrayIndices, dev_pos, dev_vel1, dev_pos_reordered, dev_vel1_reordered);
  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchCoherent << < n1, blockSize >> > (N, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos_reordered, dev_vel1_reordered, dev_vel2_reordered);
    kernUnshuffleVel2 << < n1, blockSize >> > (N, dev_particleArrayIndices, dev_vel2_reordered, dev_vel2);
    // - Update positions
    kernUpdatePos << <n1, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGridLoopingOptimization(float dt) 
{
    int n1 = (numObjects + blockSize - 1) / blockSize;
    int n2 = (gridCellCount + blockSize - 1) / blockSize;
    int N = numObjects;
    // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
    // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
    // In Parallel:
    // - Label each particle with its array index as well as its grid index.
    //   Use 2x width grids
    kernComputeIndices << < n1, blockSize >> > (N, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
    //   are welcome to do a performance comparison.
    thrust::device_ptr<int> dev_thrust_keys(dev_particleGridIndices);
    thrust::device_ptr<int> dev_thrust_values(dev_particleArrayIndices);
    thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);
    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
    kernIdentifyCellStartEnd << < n1, blockSize >> > (N, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
    //   the particle data in the simulation array.
    //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    kernShufflePosAndVel1 << < n1, blockSize >> > (N, dev_particleArrayIndices, dev_pos, dev_vel1, dev_pos_reordered, dev_vel1_reordered);
    // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchCoherentGridLoopingOptimization << < n1, blockSize >> > (N, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos_reordered, dev_vel1_reordered, dev_vel2_reordered);
    kernUnshuffleVel2 << < n1, blockSize >> > (N, dev_particleArrayIndices, dev_vel2_reordered, dev_vel2);
    // - Update positions
    kernUpdatePos << <n1, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGridSharedMemOptimization(float dt) {
    int n1 = (numObjects + blockSize - 1) / blockSize;
    int n2 = (gridCellCount + blockSize - 1) / blockSize;
    int N = numObjects;
    // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
    // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
    // In Parallel:
    // - Label each particle with its array index as well as its grid index.
    //   Use 2x width grids
    kernComputeIndices << < n1, blockSize >> > (N, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
    //checkCUDAErrorWithLine("kernComputeIndices failed!");
    // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
    //   are welcome to do a performance comparison.
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
    // - Naively unroll the loop for finding the start and end indices of each
    //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1);
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
    kernIdentifyCellStartEnd << < n1, blockSize >> > (N, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
    kernResetIntBuffer << < n2, blockSize >> > (gridCellCount, dev_gridCellPartitions, 0);
    kernIdentifyMaxNumParticlesAndPartitionsInGrid << < n2, blockSize >> > (gridCellCount, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_gridCellPartitions);
    thrust::device_ptr<int> dev_thrust_gridcellpartition(dev_gridCellPartitions);
    thrust::device_ptr<int> dev_thrust_gridcellpartitionprefixsum(dev_gridCellPartitionsPrefixSum);
    thrust::exclusive_scan(dev_thrust_gridcellpartition, dev_thrust_gridcellpartition + gridCellCount, dev_gridCellPartitionsPrefixSum);
    //checkCUDAErrorWithLine("exclusive_scan failed!");
    int b0size,lastPos,lastSize;
    int maxNumParticles;
    hipMemcpyFromSymbol(&maxNumParticles, HIP_SYMBOL(maxNumParticlesInGrid), sizeof(int));
    int nil = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(maxNumParticlesInGrid), &nil, sizeof(int));
    hipMemcpy(&lastPos, dev_gridCellPartitionsPrefixSum + gridCellCount - 1, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&lastSize, dev_gridCellPartitions + gridCellCount - 1, sizeof(int), hipMemcpyDeviceToHost);
    b0size = lastPos + lastSize;
    if (b0size > B0_size)
    {
        B0_size = b0size;
        hipFree(dev_B0start);
        hipFree(dev_B0offset);
        hipMalloc((void**)&dev_B0start, b0size * sizeof(int));
        checkCUDAErrorWithLine("hipMalloc dev_B0start failed!");
        hipMalloc((void**)&dev_B0offset, b0size * sizeof(int));
        checkCUDAErrorWithLine("hipMalloc dev_B0offset failed!");
    }
    kernCompactArray << <n2, blockSize >> > (gridCellCount, dev_gridCellPartitions, dev_gridCellPartitionsPrefixSum, dev_B0start, dev_B0offset);
    uint64_t sharedMemSize = (uint64_t)maxNumParticles * 27 * sizeof(glm::vec3) * 2;
    // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
    //   the particle data in the simulation array.
    //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    kernShufflePosAndVel1 << < n1, blockSize >> > (N, dev_particleArrayIndices, dev_pos, dev_vel1, dev_pos_reordered, dev_vel1_reordered);
    // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchCoherentSharedMemoryOptimization << < b0size, blockSize, sharedMemSize>> > (b0size, gridSideCount, maxNumParticles, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_B0start, dev_B0offset, dev_pos_reordered, dev_vel1_reordered, dev_vel2_reordered);
    checkCUDAErrorWithLine("kernUpdateVelNeighborSearchCoherentSharedMemoryOptimization failed!");
    kernUnshuffleVel2 << < n1, blockSize >> > (N, dev_particleArrayIndices, dev_vel2_reordered, dev_vel2);
    // - Update positions
    kernUpdatePos << <n1, blockSize >> > (numObjects, dt, dev_pos, dev_vel2);
    // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    std::swap(dev_vel1, dev_vel2);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_pos_reordered);
  hipFree(dev_vel1_reordered);
  hipFree(dev_vel2_reordered);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
