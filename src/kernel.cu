#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif
#define TEST2_2 0

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char* msg, int line = -1) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		if (line >= 0) {
			fprintf(stderr, "Line %d: ", line);
		}
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3* dev_pos;
glm::vec3* dev_pos1;
glm::vec3* dev_vel1;
glm::vec3* dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int* dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int* dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int* dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int* dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.

// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth;
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
	thrust::default_random_engine rng(hash((int)(index * time)));
	thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

	return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3* arr, float scale) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		glm::vec3 rand = generateRandomVec3(time, index);
		arr[index].x = scale * rand.x;
		arr[index].y = scale * rand.y;
		arr[index].z = scale * rand.z;
	}
}

__global__ void kernRearrangeArray(int N, glm::vec3* src, glm::vec3* dest, glm::vec3* src2, glm::vec3* dest2, int* indices) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		dest[index] = src[indices[index]];
		dest2[index] = src2[indices[index]];
	}
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
	numObjects = N;
	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	// LOOK-1.2 - This is basic CUDA memory management and error checking.
	// Don't forget to hipFree in  Boids::endSimulation.
	hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

	hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

	hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

	// LOOK-1.2 - This is a typical CUDA kernel invocation.
	kernGenerateRandomPosArray << <fullBlocksPerGrid, blockSize >> > (1, numObjects,
		dev_pos, scene_scale);
	checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

	// LOOK-2.1 computing grid params
#if TEST2_2
	gridCellWidth = std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
#else
	gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance);
#endif
	int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
	gridSideCount = 2 * halfSideCount;

	gridCellCount = gridSideCount * gridSideCount * gridSideCount;
	gridInverseCellWidth = 1.0f / gridCellWidth;
	float halfGridWidth = gridCellWidth * halfSideCount;
	gridMinimum.x -= halfGridWidth;
	gridMinimum.y -= halfGridWidth;
	gridMinimum.z -= halfGridWidth;

	// DONE-2.1 TODO-2.3 - Allocate additional buffers here.
	hipMalloc((void**)&dev_pos1, N * sizeof(glm::vec3));
	checkCUDAErrorWithLine("hipMalloc dev_pos1 failed!");

	hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

	hipMalloc((void**)&dev_particleGridIndices, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

	hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

	hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");
	hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3* pos, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	float c_scale = -1.0f / s_scale;

	if (index < N) {
		vbo[4 * index + 0] = pos[index].x * c_scale;
		vbo[4 * index + 1] = pos[index].y * c_scale;
		vbo[4 * index + 2] = pos[index].z * c_scale;
		vbo[4 * index + 3] = 1.0f;
	}
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3* vel, float* vbo, float s_scale) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);

	if (index < N) {
		vbo[4 * index + 0] = vel[index].x + 0.3f;
		vbo[4 * index + 1] = vel[index].y + 0.3f;
		vbo[4 * index + 2] = vel[index].z + 0.3f;
		vbo[4 * index + 3] = 1.0f;
	}
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float* vbodptr_positions, float* vbodptr_velocities) {
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

	kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_pos, vbodptr_positions, scene_scale);
	kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_vel1, vbodptr_velocities, scene_scale);

	checkCUDAErrorWithLine("copyBoidsToVBO failed!");

	hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/

/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3* pos, const glm::vec3* vel) {
	glm::vec3 my_pos = pos[iSelf];
	glm::vec3 perceived_center{ 0 };
	glm::vec3 c{ 0 };
	glm::vec3 perceived_velocity{ 0 };
	int rule1_neighbor_count = 0;
	int rule3_neighbor_count = 0;
	for (int i = 0; i < N; i++)
	{
		const glm::vec3 boid_pos = pos[i];
		if (i == iSelf) continue;
		float dist = glm::distance(boid_pos, my_pos);
		// Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
		if (dist < rule1Distance) {
			rule1_neighbor_count++;
			perceived_center += boid_pos;
		}
		// Rule 2: boids try to stay a distance d away from each other
		if (dist < rule2Distance)
			c -= (boid_pos - my_pos);
		// Rule 3: boids try to match the speed of surrounding boids
		if (dist < rule3Distance) {
			rule3_neighbor_count++;
			perceived_velocity += vel[i];
		}
	}
	glm::vec3 res = glm::vec3(0);
	if (rule1_neighbor_count > 0) {
		perceived_center /= rule1_neighbor_count;
		res += (perceived_center - my_pos) * rule1Scale;
	}
	res += c * rule2Scale;
	if (rule3_neighbor_count > 0) {
		perceived_velocity /= rule3_neighbor_count;
		res += perceived_velocity * rule3Scale;
	}
	return res;
}

/**
* DONE-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3* pos,
	glm::vec3* vel1, glm::vec3* vel2) {
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) return;
	// Compute a new velocity based on pos and vel1
	glm::vec3 vel_change = computeVelocityChange(N, index, pos, vel1);
	glm::vec3 new_vel = vel1[index] + vel_change;
	// Clamp the speed
	if (glm::length(new_vel) > maxSpeed) {
		new_vel = glm::normalize(new_vel) * maxSpeed;
	}
	// Record the new velocity into vel2. Question: why NOT vel1?
	vel2[index] = new_vel;
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3* pos, glm::vec3* vel) {
	// Update position by velocity
	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	if (index >= N) {
		return;
	}
	glm::vec3 thisPos = pos[index];
	thisPos += vel[index] * dt;

	// Wrap the boids around so we don't lose them
	thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
	thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
	thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

	thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
	thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
	thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

	pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
	return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
	glm::vec3 gridMin, float inverseCellWidth,
	glm::vec3* pos, int* indices, int* gridIndices) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// DONE-2.1
	// - Label each boid with the index of its grid cell.
	glm::vec3 curr_pos = pos[index] - gridMin;
	int x = (int)(curr_pos.x * inverseCellWidth);
	int y = (int)(curr_pos.y * inverseCellWidth);
	int z = (int)(curr_pos.z * inverseCellWidth);
	gridIndices[index] = gridIndex3Dto1D(x, y, z, gridResolution);
	// - Set up a parallel array of integer indices as pointers to the actual
	//   boid data in pos and vel1/vel2
	indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int* intBuffer, int value) {
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < N) {
		intBuffer[index] = value;
	}
}

__global__ void kernIdentifyCellStartEnd(int N, int* particleGridIndices,
	int* gridCellStartIndices, int* gridCellEndIndices) {
	// DONE-2.1
	// Identify the start point of each cell in the gridIndices array.
	// This is basically a parallel unrolling of a loop that goes
	// "this index doesn't match the one before it, must be a new cell!"
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N) return;
	if (index == 0) {
		gridCellStartIndices[particleGridIndices[index]] = index;
		return;
	}
	if (index == N - 1) {
		gridCellEndIndices[particleGridIndices[index]] = index;
		return;
	}
	if (particleGridIndices[index] != particleGridIndices[index - 1]) {
		gridCellStartIndices[particleGridIndices[index]] = index;
		gridCellEndIndices[particleGridIndices[index - 1]] = index - 1;
	}
}

__global__ void kernUpdateVelNeighborSearchScattered(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int* gridCellStartIndices, int* gridCellEndIndices,
	int* particleArrayIndices,
	glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
	// DONE-2.1 - Update a boid's velocity using the uniform grid to reduce
	// the number of boids that need to be checked.
	// - Identify the grid cell that this particle is in
	// - Identify which cells may contain neighbors. This isn't always 8.
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N)
		return;
	int iSelf = particleArrayIndices[index];
	const glm::vec3 cell_idx = (pos[iSelf] - gridMin) * inverseCellWidth;
#if TEST2_2
	const glm::ivec3 cell_idx_begin = cell_idx - glm::vec3(1);
	const glm::ivec3 cell_idx_end = cell_idx_begin + glm::ivec3(2);
#else
	const glm::ivec3 cell_idx_begin = glm::clamp(cell_idx - glm::vec3(0.5), glm::vec3(0), glm::vec3(gridResolution));
	const glm::ivec3 cell_idx_end = cell_idx_begin + glm::ivec3(2);
#endif
	const glm::vec3 my_pos = pos[iSelf];
	glm::vec3 perceived_center{ 0 };
	glm::vec3 c{ 0 };
	glm::vec3 perceived_velocity{ 0 };
	int rule1_neighbor_count = 0;
	int rule3_neighbor_count = 0;
	for (int x = cell_idx_begin.x; x < cell_idx_end.x; x++)
	{
		if (x < 0 || x >= gridResolution)
			continue;
		for (int y = cell_idx_begin.y; y < cell_idx_end.y; y++)
		{
			if (y < 0 || y >= gridResolution)
				continue;
			for (int z = cell_idx_begin.z; z < cell_idx_end.z; z++)
			{
				if (z < 0 || z >= gridResolution)
					continue;
				const int curr_grid_index = gridIndex3Dto1D(x, y, z, gridResolution);
				const int start_index = gridCellStartIndices[curr_grid_index];
				if (start_index == -1) continue;
				const int end_index = gridCellEndIndices[curr_grid_index];
				for (int i = start_index; i <= end_index; i++)
				{
					const int curr_real_id = particleArrayIndices[i];
					if (curr_real_id == iSelf) continue;
					const glm::vec3 boid_pos = pos[curr_real_id];
					const glm::vec3 vec_boid = boid_pos - my_pos;
					float dist = glm::dot(vec_boid, vec_boid);
					if (dist < rule1Distance * rule1Distance) {
						rule1_neighbor_count++;
						perceived_center += boid_pos;
					}
					if (dist < rule2Distance * rule2Distance)
						c -= vec_boid;
					if (dist < rule3Distance * rule3Distance) {
						rule3_neighbor_count++;
						perceived_velocity += vel1[curr_real_id];
					}
				}
			}
		}
	}
	glm::vec3 res = glm::vec3(0);
	if (rule1_neighbor_count > 0) {
		perceived_center /= rule1_neighbor_count;
		res += (perceived_center - my_pos) * rule1Scale;
	}
	res += c * rule2Scale;
	if (rule3_neighbor_count > 0) {
		perceived_velocity /= rule3_neighbor_count;
		res += perceived_velocity * rule3Scale;
	}
	glm::vec3 new_vel = vel1[iSelf] + res;
	float speed2 = glm::dot(new_vel, new_vel);
	if (speed2 > maxSpeed * maxSpeed) {
		new_vel = new_vel * glm::inversesqrt(speed2) * maxSpeed;
	}
	vel2[iSelf] = new_vel;
}

__global__ void kernUpdateVelNeighborSearchCoherent(
	int N, int gridResolution, glm::vec3 gridMin,
	float inverseCellWidth, float cellWidth,
	int* gridCellStartIndices, int* gridCellEndIndices,
	glm::vec3* pos, glm::vec3* vel1, glm::vec3* vel2) {
	// DONE-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
	// except with one less level of indirection.
	// This should expect gridCellStartIndices and gridCellEndIndices to refer
	// directly to pos and vel1.
	// - Identify the grid cell that this particle is in
	// - Identify which cells may contain neighbors. This isn't always 8.
	// - For each cell, read the start/end indices in the boid pointer array.
	//   DIFFERENCE: For best results, consider what order the cells should be
	//   checked in to maximize the memory benefits of reordering the boids data.
	// - Access each boid in the cell and compute velocity change from
	//   the boids rules, if this boid is within the neighborhood distance.
	// - Clamp the speed change before putting the new speed in vel2
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= N)
		return;
	int iSelf = index;
	const glm::vec3 cell_idx = (pos[iSelf] - gridMin) * inverseCellWidth;
#if TEST2_2
	const glm::ivec3 cell_idx_begin = cell_idx - glm::vec3(1);
	const glm::ivec3 cell_idx_end = cell_idx_begin + glm::ivec3(2);
#else
	const glm::ivec3 cell_idx_begin = glm::clamp(cell_idx - glm::vec3(0.5), glm::vec3(0), glm::vec3(gridResolution));
	const glm::ivec3 cell_idx_end = cell_idx_begin + glm::ivec3(2);
#endif
	const glm::vec3 my_pos = pos[iSelf];
	glm::vec3 perceived_center{ 0 };
	glm::vec3 c{ 0 };
	glm::vec3 perceived_velocity{ 0 };
	int rule1_neighbor_count = 0;
	int rule3_neighbor_count = 0;
	for (int x = cell_idx_begin.x; x < cell_idx_end.x; x++)
	{
		if (x < 0 || x >= gridResolution)
			continue;
		for (int y = cell_idx_begin.y; y < cell_idx_end.y; y++)
		{
			if (y < 0 || y >= gridResolution)
				continue;
			for (int z = cell_idx_begin.z; z < cell_idx_end.z; z++)
			{
				if (z < 0 || z >= gridResolution)
					continue;
				const int curr_grid_index = gridIndex3Dto1D(x, y, z, gridResolution);
				const int start_index = gridCellStartIndices[curr_grid_index];
				if (start_index == -1) continue;
				const int end_index = gridCellEndIndices[curr_grid_index];
				for (int i = start_index; i <= end_index; i++)
				{
					if (i == iSelf) continue;
					const glm::vec3 boid_pos = pos[i];
					const glm::vec3 vec_boid = boid_pos - my_pos;
					float dist = glm::dot(vec_boid, vec_boid);
					if (dist < rule1Distance * rule1Distance) {
						rule1_neighbor_count++;
						perceived_center += boid_pos;
					}
					if (dist < rule2Distance * rule2Distance)
						c -= vec_boid;
					if (dist < rule3Distance * rule3Distance) {
						rule3_neighbor_count++;
						perceived_velocity += vel1[i];
					}
				}
			}
		}
	}
	glm::vec3 res = glm::vec3(0);
	if (rule1_neighbor_count > 0) {
		perceived_center /= rule1_neighbor_count;
		res += (perceived_center - my_pos) * rule1Scale;
	}
	res += c * rule2Scale;
	if (rule3_neighbor_count > 0) {
		perceived_velocity /= rule3_neighbor_count;
		res += perceived_velocity * rule3Scale;
	}
	glm::vec3 new_vel = vel1[iSelf] + res;
	float speed2 = glm::dot(new_vel, new_vel);
	if (speed2 > maxSpeed * maxSpeed) {
		new_vel = new_vel * glm::inversesqrt(speed2) * maxSpeed;
	}
	vel2[iSelf] = new_vel;
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
	// DONE-1.2 - use the kernels you wrote to step the simulation forward in time.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	kernUpdateVelocityBruteForce << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_pos, dev_vel1, dev_vel2);
	kernUpdatePos << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dt, dev_pos, dev_vel2);
	// DONE-1.2 ping-pong the velocity buffers
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationScatteredGrid(float dt) {
	// DONE-2.1
	// Uniform Grid Neighbor search using Thrust sort.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	dim3 fullBlocksPerGridIntBuffer((gridCellCount + blockSize - 1) / blockSize);
	// In Parallel:
	// - label each particle with its array index as well as its grid index.
	kernComputeIndices << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	//   Use 2x width grids.
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
	dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	kernResetIntBuffer << <fullBlocksPerGridIntBuffer, threadsPerBlock >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <fullBlocksPerGridIntBuffer, threadsPerBlock >> > (gridCellCount, dev_gridCellEndIndices, -1);
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	// - Perform velocity updates using neighbor search
	kernUpdateVelNeighborSearchScattered << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);
	// - Update positions
	kernUpdatePos << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dt, dev_pos, dev_vel2);
	// - Ping-pong buffers as needed
	std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
	// TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
	// Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
	// In Parallel:
	// - Label each particle with its array index as well as its grid index.
	//   Use 2x width grids
	// - Unstable key sort using Thrust. A stable sort isn't necessary, but you
	//   are welcome to do a performance comparison.
	// - Naively unroll the loop for finding the start and end indices of each
	//   cell's data pointers in the array of boid indices
	// - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
	//   the particle data in the simulation array.
	//   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
	// - Perform velocity updates using neighbor search
	// - Update positions
	// - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
	dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
	dim3 fullBlocksPerGridIntBuffer((gridCellCount + blockSize - 1) / blockSize);
	kernComputeIndices << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);
	dev_thrust_particleGridIndices = thrust::device_ptr<int>(dev_particleGridIndices);
	dev_thrust_particleArrayIndices = thrust::device_ptr<int>(dev_particleArrayIndices);
	thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);
	kernResetIntBuffer << <fullBlocksPerGridIntBuffer, threadsPerBlock >> > (gridCellCount, dev_gridCellStartIndices, -1);
	kernResetIntBuffer << <fullBlocksPerGridIntBuffer, threadsPerBlock >> > (gridCellCount, dev_gridCellEndIndices, -1);
	kernIdentifyCellStartEnd << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);
	kernRearrangeArray << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dev_pos, dev_pos1, dev_vel1, dev_vel2, dev_particleArrayIndices);

	kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos1, dev_vel2, dev_vel1);
	kernUpdatePos << <fullBlocksPerGrid, threadsPerBlock >> > (numObjects, dt, dev_pos1, dev_vel1);
	std::swap(dev_pos, dev_pos1);
}

void Boids::endSimulation() {
	hipFree(dev_vel1);
	hipFree(dev_vel2);
	hipFree(dev_pos);

	// DONE-2.1 TODO-2.3 - Free any additional buffers here.
	hipFree(dev_pos1);
	hipFree(dev_particleArrayIndices);
	hipFree(dev_particleGridIndices);
	hipFree(dev_gridCellStartIndices);
	hipFree(dev_gridCellEndIndices);
}

void Boids::unitTest() {
	// LOOK-1.2 Feel free to write additional tests here.

	// test unstable sort
	int* dev_intKeys;
	int* dev_intValues;
	int N = 10;

	std::unique_ptr<int[]>intKeys{ new int[N] };
	std::unique_ptr<int[]>intValues{ new int[N] };

	intKeys[0] = 0; intValues[0] = 0;
	intKeys[1] = 1; intValues[1] = 1;
	intKeys[2] = 0; intValues[2] = 2;
	intKeys[3] = 3; intValues[3] = 3;
	intKeys[4] = 0; intValues[4] = 4;
	intKeys[5] = 2; intValues[5] = 5;
	intKeys[6] = 2; intValues[6] = 6;
	intKeys[7] = 0; intValues[7] = 7;
	intKeys[8] = 5; intValues[8] = 8;
	intKeys[9] = 6; intValues[9] = 9;

	hipMalloc((void**)&dev_intKeys, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

	hipMalloc((void**)&dev_intValues, N * sizeof(int));
	checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

	dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

	std::cout << "before unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// How to copy data to the GPU
	hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

	// Wrap device vectors in thrust iterators for use with thrust.
	thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
	thrust::device_ptr<int> dev_thrust_values(dev_intValues);
	// LOOK-2.1 Example for using thrust::sort_by_key
	thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

	// How to copy data back to the CPU side from the GPU
	hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
	hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
	checkCUDAErrorWithLine("memcpy back failed!");

	std::cout << "after unstable sort: " << std::endl;
	for (int i = 0; i < N; i++) {
		std::cout << "  key: " << intKeys[i];
		std::cout << " value: " << intValues[i] << std::endl;
	}

	// cleanup
	hipFree(dev_intKeys);
	hipFree(dev_intValues);
	checkCUDAErrorWithLine("hipFree failed!");
	return;
}
